#include "hip/hip_runtime.h"
#include "kernel.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "utils/cuda_helpers.h"


__global__ void kernel(float3* output, unsigned int width, unsigned int height) {
	auto x = blockIdx.x * blockDim.x + threadIdx.x;
	auto y = blockIdx.y * blockDim.y + threadIdx.y;

	auto index = (x + width * y);

	output[index].x = 1.0f;
}

void render(float3* buffer, unsigned int w, unsigned int h) {
	dim3 block(16, 16, 1);

	dim3 grid(w / block.x, h / block.y, 1);

	std::cout << "about to call kernel" << std::endl;

	kernel << <grid, block >> > (buffer, w, h);

}

__global__ void part1_kernel(uint8_t* field, uint8_t* swap_field, uint32_t* flashes, size_t width, size_t height) {


}

void part1(Managed<uint8_t>& field1, Managed<uint8_t>& field2, Managed<uint32_t>& flashed, size_t width, size_t height) {

	dim3 block(16, 16, 1);
	dim3 grid(width / block.x, height / block.y, 1);

	part1_kernel << <grid, block >> > (
		field1.get_device_ptr(),
		field2.get_device_ptr(),
		flashed.get_device_ptr(),
		width, height);
}
